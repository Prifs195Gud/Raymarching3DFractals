#include "hip/hip_runtime.h"

#include <SDF.cuh>

__device__ float SDF_Box(CVector3 point, CVector3 boxPos, CVector3 scale)
{
	CVector3 q = (point - boxPos).Absolute() - scale;
	return q.Maximum(0.0f).Magnitude() + fmin(fmax(q.x, fmax(q.y, q.z)), 0.0f);
}
__device__ float SDF_Box(CVector3 point, CVector3 scale)
{
	return SDF_Box(point, CVector3(), scale);
}

__device__ float SDF_Sphere(CVector3 point, CVector3 spherePos, float radius)
{
	return (spherePos - point).Magnitude() - radius;
}

__device__ float SDF_Sphere(CVector3 point, float radius)
{
	return point.Magnitude() - radius;
}

__device__ float SDF_Plane(CVector3 point)
{
	return point.y + 0.5f;
}


__device__ float R_SDF_Sphere(CVector3 point, float radius, float repeat)
{
	if (point.x > 0.0f)
		point.x = fmodf(point.x, repeat) - repeat * 0.5f;
	else
		point.x = -fmodf(point.x, repeat) - repeat * 0.5f;

	if (point.z > 0.0f)
		point.z = fmodf(point.z, repeat) - repeat * 0.5f;
	else
		point.z = -fmodf(point.z, repeat) - repeat * 0.5f;

	return point.Magnitude() - radius;
}

//http://blog.hvidtfeldts.net/index.php/2011/08/distance-estimated-3d-fractals-iii-folding-space/
__device__ float R_Tetrahedron(CVector3 point, int Iterations, float Scale) // ?, 10, 2.0f
{
	CVector3 a1 = CVector3(10, 10, 10);
	CVector3 a2 = CVector3(-10, -10, 10);
	CVector3 a3 = CVector3(10, -10, -10);
	CVector3 a4 = CVector3(-10, 10, -10);
	CVector3 c;

	int n = 0;
	float dist, d;

	while (n < Iterations)
	{
		c = a1;
		dist = (point - a1).Magnitude();

		d = (point - a2).Magnitude();
		if (d < dist) { c = a2; dist = d; }

		d = (point - a3).Magnitude();
		if (d < dist) { c = a3; dist = d; }

		d = (point - a4).Magnitude();
		if (d < dist) { c = a4; dist = d; }

		point = point * Scale - c * (Scale - 1.0f);
		n++;
	}

	return point.Magnitude() * powf(Scale, float(-n));
}

// Folding Space Tetrahedron
__device__ float RF_Tetrahedron(CVector3 z, int Iterations, float Scale) // ?, 10, 2.0f
{
	int n = 0;
	while (n < Iterations)
	{
		// fold 1
		if (z.x + z.y < 0)
		{
			float a = z.x;
			z.x = -z.y;
			z.y = -a;
		}

		// fold 2
		if (z.x + z.z < 0)
		{
			float a = z.x;
			z.x = -z.z;
			z.z = -a;
		}

		// fold 3	
		if (z.y + z.z < 0)
		{
			float a = z.z;
			z.z = -z.y;
			z.y = -a;
		}

		z = z * Scale - CVector3(1.f, 1.f, 1.f) * (Scale - 1.0f);

		n++;
	}

	return z.Magnitude() * powf(Scale, -float(n));
}

// https://www.shadertoy.com/view/wsVBz1
// Sierpinski Tetrahedron

// Signed distance to a tetrahedron within canonical cube
// https://www.shadertoy.com/view/Ws23zt
__device__ float SDF_Tetrahedron(CVector3 point)
{
	return (fmaxf(
		abs(point.x + point.y) - point.z,
		abs(point.x - point.y) + point.z) - 1.0f) / sqrt(3.f);
}

// Fold a point across a plane defined by a point and a normal
// The normal should face the side to be reflected
__device__ CVector3 Fold(CVector3 point, CVector3 pointOnPlane, CVector3 planeNormal)
{
	// Center plane on origin for distance calculation
	float distToPlane = (point - pointOnPlane).Scalar(planeNormal);

	// We only want to reflect if the dist is negative
	distToPlane = fminf(distToPlane, 0.0f);
	return point - planeNormal * 2.0f * distToPlane;
}

__device__ float SDF_Sierpinski(CVector3 point, int level)
{
	float scale = 1.0f;

	const CVector3 vertices[4] =
	{ CVector3(1.0, 1.0, 1.0),
		CVector3(-1.0, 1.0, -1.0),
		CVector3(-1.0, -1.0, 1.0),
		CVector3(1.0, -1.0, -1.0) };

	for (int i = 0; i < level; i++)
	{
		// Scale point toward corner vertex, update scale accumulator
		point = point - vertices[0];
		point = point * 2.0f;
		point = point + vertices[0];

		scale *= 2.0f;

		// Fold point across each plane
		for (int i = 1; i <= 3; i++)
		{
			// The plane is defined by:
			// Point on plane: The vertex that we are reflecting across
			// Plane normal: The direction from said vertex to the corner vertex
			CVector3 temp = vertices[0];
			CVector3 normal = (temp - vertices[i]).Normalize();
			point = Fold(point, vertices[i], normal);
		}
	}
	// Now that the space has been distorted by the IFS,
	// just return the distance to a tetrahedron
	// Divide by scale accumulator to correct the distance field
	return SDF_Tetrahedron(point) / scale;
}


// http://blog.hvidtfeldts.net/index.php/2011/09/distance-estimated-3d-fractals-v-the-mandelbulb-different-de-approximations/
// Mandelbulb
__device__ float Mandelbulb(CVector3 pos, int Iterations, float power, float bailout)
{
	CVector3 z = pos;
	float dr = 1.0f;
	float r = 0.0f;

	for (int i = 0; i < Iterations; i++)
	{
		r = z.Magnitude();
		if (r > bailout)
			break;

		// convert to polar coordinates
		float theta = acos(z.y / r);
		float phi = atanf(z.z / z.x);
		dr = pow(r, power - 1.0f) * power * dr + 1.0f;

		// scale and rotate the point
		float zr = pow(r, power);
		theta = theta * power;
		phi = phi * power;

		// convert back to cartesian coordinates
		z = CVector3(sin(theta) * cos(phi), cos(theta), sin(phi) * sin(theta)) * zr;
		z = z + pos;
	}

	return 0.5f * log(r) * r / dr;
}

// Menger Sponge
// https://github.com/Angramme/fractal_viewer/blob/master/fractals/menger_sponge.glsl

__device__ float Truncate(float x, float t, float s)
{
	if (abs(x) < s * 0.5f && abs(x) <= t) 
		return 0.0f;

	return x > 0.0f ? s : -s;
}

__device__ float MengerSponge(CVector3 p, int n, float scale)
{
	p = p * 1.5f;

	for (int i = 0; i < n; i++)
	{
		CVector3 ap = p.Absolute();
		float mid = fmin(ap.x, fmin(ap.y, ap.z));

		CVector3 boxP = CVector3(
			Truncate(p.x, mid, scale),
			Truncate(p.y, mid, scale),
			Truncate(p.z, mid, scale)
		);

		p = p - boxP;
		scale *= 0.33333333333333f;
	}

	scale *= 3.0f;

	return (SDF_Box(p, CVector3(scale, scale, scale))) / 1.5f;
}

__device__ float MengerSponge(CVector3 p, int n)
{
	return MengerSponge(p, n, 1.0f);
}