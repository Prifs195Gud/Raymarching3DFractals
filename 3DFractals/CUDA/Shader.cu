#include "hip/hip_runtime.h"
﻿
#include <Shader.cuh>

SDL_Color* dev_pixels = nullptr;

//https://forum.openframeworks.cc/t/hsv-color-setting/770
// H [0, 360] S and V [0.0, 1.0].
__device__ CVector3 HSVToColor(float h, float s, float v)
{
	int i = (int)floor(h / 60.0f) % 6;
	float f = h / 60.0f - floor(h / 60.0f);
	float p = v * (float)(1 - s);
	float q = v * (float)(1 - s * f);
	float t = v * (float)(1 - (1 - f) * s);
	switch (i) 
	{
	case 0: return CVector3(v, t, p);
		break;
	case 1: return CVector3(q, v, p);
		break;
	case 2: return CVector3(p, v, t);
		break;
	case 3: return CVector3(p, q, v);
		break;
	case 4: return CVector3(t, p, v);
		break;
	case 5: return CVector3(v, p, q);
	}
}

__device__ float DistanceUnion(float d1, float d2) { return min(d1, d2); }

__device__ float DistanceSubtraction(float d1, float d2) { return max(-d1, d2); }

__device__ float DistanceIntersection(float d1, float d2) { return max(d1, d2); }

__device__ float DistanceEstimator(CVector3 &point, unsigned int &scene)
{
	switch (scene)
	{
	default: // Plane
		return point.y;

	case 0:
		return SDF_Sphere(point, 1);

	case 1:
		return SDF_Box(point, CVector3(1, 1, 1));

	case 2:
		return SDF_Tetrahedron(point);

	case 4:
		return SDF_Sierpinski(point, 6);

	case 5:
		return MengerSponge(point, 6, 5);

	case 6:
		return Mandelbulb(point, 20, 8.0f, 4.0f);

	case 7: // Mandelbulb with plane
		return min(Mandelbulb(point, 20, 8.0f, 4.0f), point.y + 4.0f);

	case 8: // 3 Primitives
		float minDist = SDF_Box(point, CVector3(-2.5, 0, 0), CVector3(1, 1, 1));
		minDist = min(minDist, SDF_Sphere(point, CVector3(2.5, 0, 0), 1));
		minDist = min(minDist, SDF_Tetrahedron(point));
		return minDist;

	case 9: // Repeating spheres without a plane
		return R_SDF_Sphere(point, 0.3f, 4.0f);

	case 10: // Repeating spheres with a plane
		return min(R_SDF_Sphere(point, 0.3f, 1.0f), SDF_Plane(point));

	case 11: // Cut Mandelbulb with box
		return DistanceSubtraction(SDF_Box(point, CVector3(0.6, 0, 0), CVector3(0.5, 1.25, 1.25)), Mandelbulb(point, 20, 8.0f, 4.0f));

	case 12: // Blender comparison
		return min(Mandelbulb(point - CVector3(0, 4 ,0), 20, 8.0f, 4.0f), point.y);
	}

	return point.y;
}

__constant__ FColor skyColBottom{ 0.643f,0.858f,0.952f };
__constant__ FColor skyColTop{ 0.235f,0.552f,0.725f };
__device__ CVector3 GetSkyColor(CVector3 &dir)
{
	CVector3 colT = CVector3(skyColTop.r, skyColTop.g, skyColTop.b);
	CVector3 colB = CVector3(skyColBottom.r, skyColBottom.g, skyColBottom.b);

	dir = dir.Normalize();

	float a = dir.Scalar(CVector3(0, 1, 0)) * 2.f;

	if (a < 0.f)
		return colB;
	else if (a > 1.f)
		return colT;

	return CVector3::Lerp(colB, colT, a);
}

__device__ CVector3 GetNormal(CVector3& point, unsigned int& scene)
{
	float distance = DistanceEstimator(point, scene);
	CVector2 e = CVector2(LIGHT_NORMAL_EPSILON, 0.f);

	CVector3 n = CVector3(
		distance - DistanceEstimator(point - CVector3(e.x, e.y, e.y), scene),
		distance - DistanceEstimator(point - CVector3(e.y, e.x, e.y), scene),
		distance - DistanceEstimator(point - CVector3(e.y, e.y, e.x), scene));

	return n.Normalize();
}

//https://www.desmos.com/calculator/wcfuquljqq
__device__ float Sigmoid(float x, float multiplier, float offset)
{
	return 1 / (1 + expf(-(x * multiplier + offset)));
}

//https://iquilezles.org/articles/rmshadows/
__device__ float ShadowLevel(CVector3 &pos, CVector3 &dir, float k, unsigned int& scene)
{
	float res = 1.0;
	float ph = 1e20;

	for (float t = 0.05f; t < 1.0f; )
	{
		float h = DistanceEstimator(pos + dir * t, scene);
		if (h < 0.001)
			return 0.0;
		float y = h * h / (2.0 * ph);
		float d = sqrt(h * h - y * y);
		res = fminf(res, k * d / fmaxf(0.0, t - y));
		ph = h;
		t += h;
	}
	return res;
}

__device__ float AmbientOcclusion(CVector3 &pos, CVector3 &normal, unsigned int& scene)
{
	float acc = 0.0;

	for (int i = 1; i <= AMBIENT_OCCLUSION_SAMPLES; ++i)
	{
		float d = DistanceEstimator(pos + normal * AMBIENT_OCCLUSION_STEP * i, scene);
		acc += powf(2.0f, -i) * (i * AMBIENT_OCCLUSION_STEP - max(d, 0.0f));
	}

	return min(1.0f - AMBIENT_OCCLUSION_MULTIPLIER * acc, 1.0f);
}

__device__ float AdvancedLight(CVector3& pos, CVector3& normal, CVector3& lightDirection, unsigned int& scene)
{
	float shadow = ShadowLevel(pos, lightDirection, 10, scene);

	if (shadow < 0.05f)
		shadow = 0.0f;
	else if (shadow > 1.f)
		shadow = 1.f;

	float diffuse = lightDirection.Scalar(normal) * shadow;

	if (diffuse > 1.f)
		return 1.f;
	else if (diffuse < 0.f)
		return 0.f;

	return diffuse;
}

__device__ float SimpleLight(CVector3 &normal, CVector3 &lightDirection)
{
	float diffuse = lightDirection.Scalar(normal);

	if (diffuse > 1.f)
		return 1.f;
	else if (diffuse < 0.f)
		return 0.f;

	return diffuse;
}

__device__ float RayMarch(float rayOffset, float &AO, CVector3 from, CVector3 direction, int &steps, SceneParameters& params)
{
	float totalDistance = rayOffset;
	for (steps = 0; steps < params.maxRaySteps; steps++)
	{
		CVector3 point = from + direction * totalDistance;

		float distance = fabsf(DistanceEstimator(point, params.scene));
		totalDistance += distance;

		if (distance < MIN_DISTANCE || distance > MAX_DISTANCE)
			break;
	}

	AO = 1.0f - float(steps) / float(MAX_RAY_STEPS);
	AO = Clamp(powf(AO, 2.0f), 0, 1);
	return totalDistance;
}

__device__ CVector3 GetRayDir(int &screenCoordX, int &screenCoordY, SceneParameters &params)
{
	CVector2 screenCoord = CVector2(screenCoordX, screenCoordY);

	CVector2 iResolution = CVector2(params.width, params.height);

	CVector2 uv = (screenCoord - iResolution * 0.5f) / iResolution.y;

	CVector3 rayDir = CVector3(uv.x, -uv.y, CAMERA_FOV).Normalize();

	rayDir = CVector3::RotateByX(rayDir, params.rx);
	rayDir = CVector3::RotateByY(rayDir, params.ry);

	return rayDir;
}

__device__ float GetLight(float &dist, float& reflectRatio, float &specular, CVector3 &reflectSkyCol, CVector3 &rayDir, CVector3 &point, SceneParameters &params)
{
	/*if (dist - 1.f > MAX_DISTANCE)
		return 0.f;

	return 1 - (dist / 10.0f);*/

	CVector3 normal = GetNormal(point, params.scene);

	CVector3 lightDir = CVector3(0.0f, 0.0f, 1.0f);
	lightDir = CVector3::RotateByX(lightDir, params.lrx);
	lightDir = CVector3::RotateByY(lightDir, params.lry);
	lightDir = lightDir.Normalize();

	float light = 1.f;

	if (params.lightingMode && dist < MAX_LIGHT_DISTANCE)
		light = AdvancedLight(point, normal, lightDir, params.scene);
		//light = SimpleLight(normal, lightDir);
	light = pow(light, 0.45454545454f); // Gamma light level correction

	specular = powf(CVector3::Similarity(lightDir, normal), 25.0f);
	reflectSkyCol = GetSkyColor(CVector3::Reflect(rayDir, normal));
	reflectRatio = Clamp01(1.0f - CVector3::Similarity(-rayDir, normal) * 0.5f);

	return light;
}

// https://blog.demofox.org/2020/05/10/ray-marching-fog-with-blue-noise/
// https://www.shadertoy.com/view/WsfBDf
__device__ float InterleavedGradientNoise(float screenCoordX, float screenCoordY)
{
	float a = 0.06711056f * screenCoordX + 0.00583715f * screenCoordY;
	a = fmodf(a, 1);
	a = 52.9829189f * a;
	a = fmodf(a, 1);
	return a;
}

__device__ CVector3 GetColor(int& index, SceneParameters& params)
{
	int screenCoordY = index / params.width;
	int screenCoordX = index - params.width * screenCoordY;

	CVector3 rayDir = GetRayDir(screenCoordX, screenCoordY, params);
	CVector3 cam = CVector3(params.cx, params.cy, params.cz);

	int steps;

	float AO, rayOffset = InterleavedGradientNoise(screenCoordX, screenCoordY) * 0.01f;

	if (!params.ditherMode)
		rayOffset = 0;

	float dist = RayMarch(rayOffset, AO, cam, rayDir, steps, params);
	//AO = Clamp(AO, 0.5f, 1.0f);

	CVector3 skyColor = GetSkyColor(rayDir);
	if (dist - 1.f > MAX_DISTANCE)
		return skyColor;

	if (params.heatmapMode)
		return HSVToColor((1 - (steps / (float)MAX_RAY_STEPS)) * 243.0f, 1, 1);

	float specular, reflectRatio;
	float fog = (1 / exp2f(FOG_DENSITY * dist)); // Double exp fog
	CVector3 reflectSkyCol;

	CVector3 point = cam + rayDir * dist;
	float light = GetLight(dist, reflectRatio, specular, reflectSkyCol, rayDir, point, params);

	CVector3 objectCol = CVector3(1.0f, 1.0f, 1.0f);
	CVector3 col = CVector3::Lerp(objectCol, reflectSkyCol, Clamp(reflectRatio, 0.5f, 0.75f)); // Ambient light
	col = CVector3::Lerp(col, objectCol, light); // Object color
	col = CVector3::Lerp(col, reflectSkyCol * 0.1f, 1.0f - Clamp(light, 0.1f, 1.0f)); // Add shadows

	if(params.AOMode)
		col = col * Lerp(AO, 1, reflectRatio); // Add ambient occlusion

	col = CVector3::Lerp(col, skyColor, 1 - fog); // Add fog

	return col;
}

__global__ void Shader(SceneParameters params)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//int index = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

	if (index >= params.width * params.height) // Offscreen
		return;

	CVector3 col = GetColor(index, params);

	if (col.x < 0.f)
		col.x = 0.f;
	else if (col.x > 1.f)
		col.x = 1.f;

	if (col.y < 0.f)
		col.y = 0.f;
	else if (col.y > 1.f)
		col.y = 1.f;

	if (col.z < 0.f)
		col.z = 0.f;
	else if (col.z > 1.f)
		col.z = 1.f;

	SDL_Color* pixel = &params.pixelBuffer[index];
	pixel->r = (int)(col.z * 255);
	pixel->g = (int)(col.y * 255);
	pixel->b = (int)(col.x * 255);
}

int lastWidth = 0, lastHeight = 0;
void Initialize(int width, int height)
{
	lastWidth = width;
	lastHeight = height;
	CudaMalloc(dev_pixels, sizeof(SDL_Color) * width * height);
}

void FreeVideoMemory()
{
	hipFree(dev_pixels);
	CudaErrorCheck();
}

void ApplyShader(SceneParameters params, bool copyPixelsFromGPU)
{
	int totalPixels = params.width * params.height;

	//int sqrThreads = (int)lround(sqrt(params.threadsPerBlock)); // 2D mode

	int blockCount = (int)ceil((float)totalPixels / (float)params.threadsPerBlock);

	if (dev_pixels == nullptr || params.width != lastWidth || params.height != lastHeight)
	{
		if(dev_pixels != nullptr)
			FreeVideoMemory();
		Initialize(params.width, params.height);
		CudaErrorCheck();
	}

	SceneParameters shaderParams = params;
	shaderParams.pixelBuffer = dev_pixels;

	dim3 threadsPerBlock(params.threadsPerBlock, 1, 1); // 1D mode
	//dim3 threadsPerBlock(sqrThreads, sqrThreads, 1); // 2D mode
	Shader <<<blockCount, threadsPerBlock >>> (shaderParams);
	CudaErrorCheck();

	CudaDeviceSynchronize();
	CudaErrorCheck();

	if (copyPixelsFromGPU)
	{
		CudaCopyFromGPU(params.pixelBuffer, dev_pixels, sizeof(SDL_Color) * totalPixels);
		CudaErrorCheck();
	}
}