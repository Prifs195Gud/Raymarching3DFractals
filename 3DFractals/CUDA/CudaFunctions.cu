
#include <CudaFunctions.cuh>

void SendErrorLog(string str)
{
	cout << str << '\n';
}

string CudaGetErrorString(hipError_t errCode)
{
	return string(hipGetErrorString(errCode));
}

bool CudaSetDevice()
{
	hipError_t err = hipSetDevice(0);

	if (err != hipSuccess)
		SendErrorLog("hipSetDevice failed! Do you have a CUDA-capable GPU installed? " + CudaGetErrorString(err));

	return err == hipSuccess;
}

bool CCudaMalloc(void** devPtr, size_t size)
{
	hipError_t err = hipMalloc(devPtr, size);

	if (err != hipSuccess)
		SendErrorLog("hipMalloc failed! " + CudaGetErrorString(err));

	return err == hipSuccess;
}

bool CudaDeviceSynchronize()
{
	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess)
		SendErrorLog("hipDeviceSynchronize failed! " + CudaGetErrorString(err));

	return err == hipSuccess;
}

bool CudaCopyFromGPU(void* host_ptr, void* gpu_ptr, size_t size)
{
	hipError_t err = hipMemcpy(host_ptr, gpu_ptr, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
		SendErrorLog("hipMemcpy from device failed! " + CudaGetErrorString(err));

	return err == hipSuccess;
}

bool CudaCopyToGPU(void* host_ptr, void* gpu_ptr, size_t size)
{
	hipError_t err = hipMemcpy(gpu_ptr, host_ptr, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
		SendErrorLog("hipMemcpy to device failed! " + CudaGetErrorString(err));

	return err == hipSuccess;
}

void CudaErrorCheck()
{
	hipError_t err = hipGetLastError();

	if (err == hipSuccess)
		return;

	SendErrorLog("Cuda error! " + CudaGetErrorString(err));
}